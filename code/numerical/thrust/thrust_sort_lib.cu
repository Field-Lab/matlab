/**
 * To compile:
 * /home/peterli/cuda-4.0.17/bin/nvcc -I/home/peterli/cuda-4.0.17/include -c thrust_sort_lib.cu -Xcompiler -fPIC
 *
 * Note, CUDA toolset version must match that included in Matlab if this is to
 * be called from Matlab Mex.
 *
 * -fPIC needed for call from Matlab Mex as well.
 *
 */

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include "mx_functional/skip_ptr.h"


/**
 * thrust_device_sort
 */
template<typename T>
void thrust_device_sort(T first, T last) {
  typedef typename T::value_type V;
  thrust::device_vector<V> dv(first, last);
  thrust::sort(dv.begin(), dv.end());
  thrust::copy(dv.begin(), dv.end(), first);
}

// Explicit instantiations
template void thrust_device_sort<skip_ptr<double> >(skip_ptr<double>, skip_ptr<double>);
template void thrust_device_sort<skip_ptr<float> >(skip_ptr<float>, skip_ptr<float>);
template void thrust_device_sort<skip_ptr<int> >(skip_ptr<int>, skip_ptr<int>);
template void thrust_device_sort<skip_ptr<unsigned int> >(skip_ptr<unsigned int>, skip_ptr<unsigned int>);
template void thrust_device_sort<skip_ptr<short> >(skip_ptr<short>, skip_ptr<short>);
template void thrust_device_sort<skip_ptr<unsigned short> >(skip_ptr<unsigned short>, skip_ptr<unsigned short>);
template void thrust_device_sort<skip_ptr<char> >(skip_ptr<char>, skip_ptr<char>);
template void thrust_device_sort<skip_ptr<unsigned char> >(skip_ptr<unsigned char>, skip_ptr<unsigned char>);



/**
 * thrust_sort
 */
template<typename T>
void thrust_sort(T first, T last) {
  thrust::sort(first, last);
}

// Explicit instantiations
template void thrust_sort<skip_ptr<double> >(skip_ptr<double>, skip_ptr<double>);
template void thrust_sort<skip_ptr<float> >(skip_ptr<float>, skip_ptr<float>);
template void thrust_sort<skip_ptr<int> >(skip_ptr<int>, skip_ptr<int>);
template void thrust_sort<skip_ptr<unsigned int> >(skip_ptr<unsigned int>, skip_ptr<unsigned int>);
template void thrust_sort<skip_ptr<short> >(skip_ptr<short>, skip_ptr<short>);
template void thrust_sort<skip_ptr<unsigned short> >(skip_ptr<unsigned short>, skip_ptr<unsigned short>);
template void thrust_sort<skip_ptr<char> >(skip_ptr<char>, skip_ptr<char>);
template void thrust_sort<skip_ptr<unsigned char> >(skip_ptr<unsigned char>, skip_ptr<unsigned char>);
